#include "hip/hip_runtime.h"
#include "forward.h"
#include "matrix.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include <stdbool.h>
#include <stdint.h>
#include <assert.h>

__global__ void forwardGPU(double *A, double *B, double *C, double *D, double *res1, double *res2,
                           int numARows, int numAColumns,
                           int numBRows, int numBColumns,
                           int numCRows, int numCColumns,
                           int numDRows, int numDColumns)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numARows && col < numBColumns)
    {
        float sumAB = 0;
        float sumCD = 0;
        int idx = row * numBColumns + col;
        for (int ii = 0; ii < numAColumns; ii++)
        {
            sumAB += A[row * numAColumns + ii] * B[ii * numBColumns + col];
        }
        for (int ii = 0; ii < numCColumns; ii++)
        {
            sumCD += C[row * numCColumns + ii] * D[ii * numDColumns + col];
        }
        res1[idx] = sumAB + sumCD;
        res2[idx] = 1 / (1 + exp(-res1[idx]));
        }
}

void forward_operations(ann_t *nn, int l)
{

    // allocations
    // matrix_t *z1 = alloc_matrix(nn->layers[l]->number_of_neurons, nn->minibatch_size);
    // matrix_t *z2 = alloc_matrix(nn->layers[l]->number_of_neurons, nn->minibatch_size);
    matrix_t *one = alloc_matrix(1, nn->minibatch_size);
    // init one to ones
    for (int idx = 0; idx < one->columns * one->rows; idx++)
        one->m[idx] = 1.0;

    // check dimensions for w^l x a^(l-1)
    assert(nn->layers[l]->weights->columns == nn->layers[l - 1]->activations->rows);

    // check dimensions for b^l x 1
    assert(nn->layers[l]->biases->columns == one->rows);

    // check dimensions for z^l = w^l x a^(l-1) + b^l x 1
    assert((nn->layers[l]->weights->rows == nn->layers[l]->biases->rows) &&
           (nn->layers[l - 1]->activations->columns == nn->layers[l]->z->columns) &&
           (nn->layers[l - 1]->activations->columns == one->columns) &&
           (nn->layers[l]->weights->rows == nn->layers[l]->z->rows));
    // check dimensions for f(z^l)
    assert((nn->layers[l]->z->columns == nn->layers[l]->activations->columns) &&
           (nn->layers[l]->z->rows == nn->layers[l]->activations->rows));

    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)nn->layers[l]->activations->columns) / blockDim.x), ceil(((float)nn->layers[l]->activations->rows) / blockDim.y));
    forwardGPU<<<gridDim, blockDim>>>(nn->layers[l]->weights->m, nn->layers[l - 1]->activations->m,
                                      nn->layers[l]->biases->m, one->m,
                                      nn->layers[l]->z->m, nn->layers[l]->activations->m,
                                      nn->layers[l]->weights->rows, nn->layers[l]->weights->columns,
                                      nn->layers[l - 1]->activations->rows, nn->layers[l - 1]->activations->columns,
                                      nn->layers[l]->biases->rows, nn->layers[l]->biases->columns,
                                      one->rows, one->columns);

    hipDeviceSynchronize();

    destroy_matrix(one);
}